#include "hip/hip_runtime.h"
// Copyright 2023. All Rights Reserved.
// Author: Bruce-Lee-LY
// Date: 21:02:28 on Tue, Feb 28, 2023
//
// Description: mma base hgemm



// this is pipeline optimization base code, please read the mma_async.cu,mma_async.stage3.cu in order 
// the pipeline optimization descrided in  https://zhuanlan.zhihu.com/p/665082713 in detail 
#include "common.h"

#define MMA_M 16
#define MMA_N 8
#define MMA_K 16 //每个warp负责的mma计算size.

//Block Tile
#define BLOCK_ROWS 256 //每个BLOCK负责D矩阵的多少行生成
#define BLOCK_COLS 128 //每个Block负责D矩阵的多少列生成
//Warp Tile
#define WARP_ROWS 64   //每个WARP负责D矩阵的多少行的生成
#define WARP_COLS 64   //每个warp负责D矩阵的多少列生成

//Block Dim ,每行有BLOCK_ROW_WARPS,每列有BLOCK_COL_WARPS
#define BLOCK_ROW_WARPS 2  // BLOCK_COLS / WARP_COLS  
#define BLOCK_COL_WARPS 4  // BLOCK_ROWS / WARP_ROWS

// 使用Tensor Core时需要对BLOCK Tile进行Tensor Core的MMA size来进行 Tile.
// 即BLOCK Tile中每行需要BLOCK_ROW_TILES次进行mma计算，每列需要BLOCK_COL_TILES_次mma计算
#define BLOCK_ROW_TILES 16  // BLOCK_COLS / MMA_N
#define BLOCK_COL_TILES 16  // BLOCK_ROWS / MMA_M
// 使用Tensor Core时需要对Warp 
#define WARP_ROW_TILES 8  // WARP_COLS / MMA_N
#define WARP_COL_TILES 4  // WARP_ROWS / MMA_M

#define WARP_SIZE 32
#define WARPS_PER_BLOCK 8      // BLOCK_ROW_WARPS * BLOCK_COL_WARPS
#define THREADS_PER_BLOCK 256  // WARP_SIZE * WARPS_PER_BLOCK


// chunk: tensor core中4*8矩阵或8*4矩阵称为一个chunk.chunk_k表示k维度上的chunk数.
#define CHUNK_K 2  // 32 / MMA_K 

#define CHUNK_LINE_BYTES 64          // CHUNK_K * MMA_K * sizeof(half) :每个MMA_K的元素类型为half. 64字节是4个int4
#define CHUNK_COPY_LINES_PER_WARP 8  // WARP_SIZE * sizeof(int4) / CHUNK_LINE_BYTES: int4表示4个int值组成,因此一个CHUNK_LINE可以排放4个threads
#define CHUNK_COPY_LINE_LANES 4      // WARP_SIZE / CHUNK_COPY_LINES_PER_WARP

#define AB_SMEM_STRIDE 32  // CHUNK_K * MMA_K

#define C_SMEM_STRIDE 128  // BLOCK_COLS  因为一个BLOCK是由BLOCK_ROW_WARP*BLOCK_COL_WARP组成,因此STRIDE与OFFSET不一致.
#define C_SMEM_OFFSET 64   // WARP_COLS

#define BLOCK_STRIDE 16
// A: M*K, B: N*K
__global__ void mmaBaseKernel(const half *__restrict__ A, const half *__restrict__ B, half *__restrict__ C, size_t M,
                              size_t N, size_t K) {
    const size_t M_tiles = div_ceil(M, MMA_M);
    const size_t N_tiles = div_ceil(N, MMA_N);
    const size_t K_tiles = div_ceil(K, MMA_K);
    //把Grid进行展开 block index映射到 mma tensor core的排序中
    const size_t block_tile_i = 
        (blockIdx.z % 2) ? ((gridDim.y - blockIdx.y - 1) * BLOCK_COL_TILES) : (blockIdx.y * BLOCK_COL_TILES); //TODO: 我感觉是blockIdx.y * BLOCK_ROW_TILES?
    const size_t block_tile_j = (blockIdx.z * gridDim.x + blockIdx.x) * BLOCK_ROW_TILES; // TODO: 我感觉是 (blockIdx.z * gridDim.x + blockIdx.x) *BLOCK_COL_TILES?

    if (block_tile_i >= M_tiles || block_tile_j >= N_tiles) {
        return;
    }
    //1. smem保存的是A,B矩阵
    extern __shared__ half smem[][AB_SMEM_STRIDE];  //AB_SMEM_STRIDE: CHUNK_K * MMA_K =2*16=32
    // 计算出当前thread得warp_id以及lane_id
    const size_t warp_id = threadIdx.x / WARP_SIZE; 
    const size_t lane_id = threadIdx.x % WARP_SIZE;

    constexpr size_t B_smem_idx_off = BLOCK_ROWS;
    //2. 计算当前线程所在的warp tile在C的shared memory的开始位置.
    half *smem_warp_tile_row_ptr = &smem[0][0] + (warp_id / BLOCK_ROW_WARPS) * C_SMEM_STRIDE * WARP_ROWS; 
    const half *smem_warp_stream_ptr = &smem[0][0] + warp_id * MMA_M * 2 * C_SMEM_STRIDE; 
    //3. 当前thread所在的warp 在C矩阵中的读取元素开始位置
    const size_t gmem_idx = (block_tile_i + warp_id * 2) * MMA_M * N + block_tile_j * MMA_N;//TODO: warp*2的原因
    const half *src_gmem_warp_stream_ptr = &C[gmem_idx];

    uint32_t RC[WARP_COL_TILES][WARP_ROW_TILES][2]; //当前线程所在warp的mma的计算结果小c. 

//初始化
#pragma unroll
    for (size_t i = 0; i < WARP_COL_TILES; ++i) {
#pragma unroll
        for (size_t j = 0; j < WARP_ROW_TILES; ++j) {
            RC[i][j][0] = 0;
            RC[i][j][1] = 0;
        }
    }
    // 这个地方需要把WARPS_PER_BLOCK 拆成 BLOCK_ROW_WARPS * BLOCK_COL_WARPS : 
    // BLOCK_ROWS / WARPS_PER_BLOCK * K * warp_id=> BLOCK_ROWS / BLOCK_ROW_WARPS[WARPS_ROWS] * K * warp_id/BLOCK_COL_WARPS;
    const half *A_warp_ptr = &A[block_tile_i * MMA_M * K] + BLOCK_ROWS / WARPS_PER_BLOCK * K * warp_id;  
    const half *B_warp_ptr = &B[block_tile_j * MMA_N * K] + BLOCK_COLS / WARPS_PER_BLOCK * K * warp_id;

    constexpr size_t A_smem_iters = BLOCK_ROWS / (CHUNK_COPY_LINES_PER_WARP * WARPS_PER_BLOCK);
    constexpr size_t B_smem_iters = BLOCK_COLS / (CHUNK_COPY_LINES_PER_WARP * WARPS_PER_BLOCK);
// k_slices
#pragma unroll
    for (size_t tile_k = 0; tile_k < K_tiles; tile_k += CHUNK_K) { // K_tiles=K/MNA_K;   CHUNK_K=2
        size_t A_smem_idx = BLOCK_ROWS / WARPS_PER_BLOCK * warp_id;
        int4 *A_lane_ptr = (int4 *)(A_warp_ptr + tile_k * MMA_K + (lane_id / CHUNK_COPY_LINE_LANES) * K) +
                           (lane_id % CHUNK_COPY_LINE_LANES);
        A_smem_idx += lane_id / CHUNK_COPY_LINE_LANES;

#pragma unroll
        for (size_t i = 0; i < A_smem_iters; ++i) {
            *((int4 *)&smem[A_smem_idx][0] + (lane_id % CHUNK_COPY_LINE_LANES)) = *A_lane_ptr;

            A_lane_ptr = (int4 *)((half *)A_lane_ptr + CHUNK_COPY_LINES_PER_WARP * K);
            A_smem_idx += CHUNK_COPY_LINES_PER_WARP;
        }

        size_t B_smem_idx = B_smem_idx_off + BLOCK_COLS / WARPS_PER_BLOCK * warp_id;
        int4 *B_lane_ptr = (int4 *)(B_warp_ptr + tile_k * MMA_K + (lane_id / CHUNK_COPY_LINE_LANES) * K) +
                           (lane_id % CHUNK_COPY_LINE_LANES);
        B_smem_idx += lane_id / CHUNK_COPY_LINE_LANES;

#pragma unroll
        for (size_t i = 0; i < B_smem_iters; ++i) {
            *((int4 *)&smem[B_smem_idx][0] + (lane_id % CHUNK_COPY_LINE_LANES)) = *B_lane_ptr;

            B_lane_ptr = (int4 *)((half *)B_lane_ptr + CHUNK_COPY_LINES_PER_WARP * K);
            B_smem_idx += CHUNK_COPY_LINES_PER_WARP;
        }

        __syncthreads(); //上述的代码是将global memory->shared memory

#pragma unroll
        for (size_t k_step = 0; k_step < CHUNK_K; ++k_step) {
            uint32_t RA[WARP_COL_TILES][4];
            uint32_t RB[WARP_ROW_TILES][2];

#pragma unroll
            for (size_t i = 0; i < WARP_COL_TILES; ++i) {
                size_t A_smem_idx = (warp_id / BLOCK_ROW_WARPS) * WARP_ROWS + i * MMA_M;
                uint32_t A_smem_lane_addr =
                    __cvta_generic_to_shared(&smem[A_smem_idx + lane_id % 16][k_step * MMA_K + (lane_id / 16) * 8]);

                LDMATRIX_X4(RA[i][0], RA[i][1], RA[i][2], RA[i][3], A_smem_lane_addr); //这个是shared memory->register 【sync】
            }

#pragma unroll
            for (size_t j = 0; j < WARP_ROW_TILES; ++j) {
                size_t B_smem_idx = B_smem_idx_off + (warp_id % BLOCK_ROW_WARPS) * WARP_COLS + j * MMA_N;
                uint32_t B_smem_lane_addr =
                    __cvta_generic_to_shared(&smem[B_smem_idx + lane_id % 8][k_step * MMA_K + ((lane_id / 8) % 2) * 8]);

                LDMATRIX_X2(RB[j][0], RB[j][1], B_smem_lane_addr); //这个是shared memory->register [sync]
            }

#pragma unroll
            for (size_t i = 0; i < WARP_COL_TILES; ++i) {
#pragma unroll
                for (size_t j = 0; j < WARP_ROW_TILES; ++j) {
                    size_t j_s = (i % 2) ? (WARP_ROW_TILES - j - 1) : j;

                    HMMA16816(RC[i][j_s][0], RC[i][j_s][1], RA[i][0], RA[i][1], RA[i][2], RA[i][3], RB[j_s][0], //m:16,n:8,k:16
                              RB[j_s][1], RC[i][j_s][0], RC[i][j_s][1]); //这个是mma计算 [sync]
                }
            }
        }

        __syncthreads();
    }

// Register to Global Memory
#pragma unroll
    for (size_t i = 0; i < WARP_COL_TILES; ++i) {
#pragma unroll
        for (size_t j = 0; j < WARP_ROW_TILES; ++j) {
            half *lane_ptr0 = smem_warp_tile_row_ptr + (i * MMA_M + lane_id / 4) * C_SMEM_STRIDE +
                              (warp_id % BLOCK_ROW_WARPS) * C_SMEM_OFFSET + j * MMA_N +
                              (lane_id % 4) * sizeof(uint32_t) / sizeof(half);
            half *lane_ptr1 = smem_warp_tile_row_ptr + (i * MMA_M + lane_id / 4 + 8) * C_SMEM_STRIDE +
                              (warp_id % BLOCK_ROW_WARPS) * C_SMEM_OFFSET + j * MMA_N +
                              (lane_id % 4) * sizeof(uint32_t) / sizeof(half);

            *((uint32_t *)(lane_ptr0)) = RC[i][j][0];
            *((uint32_t *)(lane_ptr1)) = RC[i][j][1];
        }
    }

    __syncthreads();

#pragma unroll
    for (size_t i = 0; i < MMA_M; ++i) {
        *((int4 *)(src_gmem_warp_stream_ptr + (i * 2 + lane_id / 16) * N) + lane_id % 16) =
            *((int4 *)(smem_warp_stream_ptr + (i * 2 + lane_id / 16) * C_SMEM_STRIDE) + lane_id % 16);
    }
}

size_t initMmaBase() {
    int dev_id = 0;
    HGEMM_CHECK_CUDART_ERROR(hipGetDevice(&dev_id));

    hipDeviceProp_t dev_prop;
    HGEMM_CHECK_CUDART_ERROR(hipGetDeviceProperties(&dev_prop, dev_id));

    size_t smem_max_size =
        std::max((BLOCK_ROWS + BLOCK_COLS) * AB_SMEM_STRIDE * sizeof(half), BLOCK_ROWS * C_SMEM_STRIDE * sizeof(half));
    HLOG("smem_max_size: %.0f KBytes (%zu Bytes)", static_cast<double>(smem_max_size) / 1024, smem_max_size);

    HGEMM_CHECK_GT(dev_prop.sharedMemPerMultiprocessor, smem_max_size);
    HGEMM_CHECK_CUDART_ERROR(
        hipFuncSetAttribute(reinterpret_cast<const void*>(mmaBaseKernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_max_size));

    return smem_max_size;
}

void mmaBase(half *A, half *B, half *C, size_t M, size_t N, size_t K) {
    static size_t smem_max_size = initMmaBase();

    dim3 block(THREADS_PER_BLOCK);
    dim3 grid(BLOCK_STRIDE, div_ceil(M, BLOCK_ROWS), div_ceil(N, BLOCK_COLS * BLOCK_STRIDE)); // N被BLOCK_COLS*BLOCK_STRIDE去划分

    mmaBaseKernel<<<grid, block, smem_max_size>>>(A, B, C, M, N, K);
}
